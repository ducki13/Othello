
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

#define OCCUPIED(board, field) ((board) & (1L<<(field)))
#define ON_BOARD(field) (0 <= (field) && (field) < 64)
#define EVALUATE(p1, p2) ((builtin_popcount(p1))-(builtin_popcount(p2)))

extern "C" {

const int INF = 128;
const int BOARD_SIZE = 8;
const int WARP_SIZE = 32;
const int MAX_DEPTH = 1;
const int STACK_SIZE = MAX_DEPTH * BOARD_SIZE * BOARD_SIZE;

__device__ void print(long long int X) {
    printf(">>%lld\n", X);
    for (int i = 0; i < BOARD_SIZE; ++i) {
        for (int j = 0; j < BOARD_SIZE; ++j) {
            if (X & (1L << (i * BOARD_SIZE + j))) printf("1");
            else printf("0");
        }
        printf("\n");
    }
}

__device__ int builtin_popcount(long long int x) {
	int ret = 0;
	for(int i = 0; i < 64; ++i) {
		if(x&(1L<<i)) ++ret;
	}
	return ret;
}

/* args: Boards[N][2], N-count of boards, PossibleMoves[N][64], Results[N], player_token-'O' or 'X' */
__global__ void generate_moves(long long *Boards, int N, long long *PossibleMoves, int *Results) {
   
	int X[8] = {-1, -1, 0, 1, 1, 1, 0, -1};
 	int Y[8] = {0, -BOARD_SIZE, -BOARD_SIZE, -BOARD_SIZE, 0, BOARD_SIZE, BOARD_SIZE, BOARD_SIZE};

    // 28KB
    __shared__ long long int S[4][STACK_SIZE][2];
    __shared__ int Result[4][STACK_SIZE];
    __shared__ int Parent[4][STACK_SIZE];
    __shared__ int Depth[4][STACK_SIZE];
    __shared__ int Size[4];

    /* INDICES */
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int board_id = thread_id / WARP_SIZE;
    if ( board_id >= N ) return;
    int field_id = (int) threadIdx.x % WARP_SIZE;
    int idx_board = board_id % 4;

    /* COPY INPUT BOARD */
    if (field_id == 0) {
        S[idx_board][0][0] = Boards[board_id];
        S[idx_board][0][1] = Boards[N + board_id];
        Depth[idx_board][0] = MAX_DEPTH;
        Parent[idx_board][0] = -1;
        Result[idx_board][0] = -INF;
        Size[idx_board] = 1;
    }
    __syncthreads();

    /* DFS */
    while (Size[idx_board] != 0) {
        int end = Size[idx_board] - 1;
        long long player_pawns = S[idx_board][end][0];
        long long opponent_pawns = S[idx_board][end][1];
        int parent = Parent[idx_board][end];
        int depth = Depth[idx_board][end];
        bool pop_vertex = false;
		
        if (depth == 0 && field_id == 0) Result[idx_board][end] = EVALUATE(player_pawns, opponent_pawns);	
        __syncthreads();

        // terminal
        if (field_id == 0 && Result[idx_board][end] != -INF) {												
            if(parent != -1) Result[idx_board][parent] = max(Result[idx_board][parent], -Result[idx_board][end]);			
            Size[idx_board] -= 1;
            pop_vertex = true;																			
        }
        __syncthreads();

        // visit current node
        if (field_id == 0 && !pop_vertex) Result[idx_board][end] = EVALUATE(player_pawns, opponent_pawns);

        // thread #idInBoard processes fields idInBoard and idInBoard + 32
        if(!pop_vertex)
        for (int k = 0; k < 2; ++k) {
            int field = WARP_SIZE * k + field_id;

            // Move cannot be applied if the field is occupied
            if (!OCCUPIED(player_pawns, field) && !OCCUPIED(opponent_pawns, field)) {    //check whether field is free
                bool flag = false;
                long long tmp = 0;

                // Try all 8 directions
                for (int i = 0; i < 8; ++i) {
                    // Direction of the move
                    int shift = X[i] + Y[i];
                    int opponents_field = field + shift;
                    int j = 0;

                    // Continue as long as fields in the row are occupied by the opponent
                    while (ON_BOARD(opponents_field) && OCCUPIED(opponent_pawns, opponents_field)) {
                        j++;
                        opponents_field += shift;
                    }

                    // If move is possible, Reversi!
                    if (ON_BOARD(opponents_field) && OCCUPIED(player_pawns, opponents_field) && j > 1) {
                        // Reversi!
                        while (opponents_field != field) {
                            tmp |= 1L << opponents_field;    //all gained fields
                            opponents_field -= shift;
                        }
                        flag = true;
                    }
                }

                // Place the new pawn
                tmp |= 1L << field;
                // Avoid if
                tmp *= flag;
                PossibleMoves[board_id * 64 + field] = tmp;            //save gained fields
            }
        }
        __syncthreads();

        //zero-thread in board pushes possible moves onto stack
        if (field_id == 0 && !pop_vertex) {
            for (int i = 0; i < BOARD_SIZE; ++i) {
                for (int j = 0; j < BOARD_SIZE; ++j) {
                    long long move = PossibleMoves[board_id * 64 + i * BOARD_SIZE + j];
                    if (move) {
                        int top = Size[idx_board];
                        Size[idx_board] = top + 1;				
                        S[idx_board][top][1] = player_pawns | move;        //old fields + gained
                        S[idx_board][top][0] = opponent_pawns ^ move;      //old fields xor gained by opponent player
                        Parent[idx_board][top] = end;
                        Result[idx_board][top] = -INF;
                        Depth[idx_board][top] = depth - 1;
                    }
                }
            }
        }
        __syncthreads();
    }
    Results[board_id] = Result[idx_board][0];
}

}
